#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include "tensor_ops.h"

// Helper function to get the number of elements in a tensor
__device__ __host__ size_t get_num_elements(const int* shape, int ndim) {
    size_t size = 1;
    for (int i = 0; i < ndim; i++) {
        size *= shape[i];
    }
    return size;
}

// Helper function to calculate total size
size_t get_total_size(const std::vector<size_t>& shape) {
    size_t size = 1;
    for (size_t dim : shape) {
        size *= dim;
    }
    return size;
}

// CUDA kernel for element-wise addition
__global__ void add_kernel(const float* a, const float* b, float* out, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] + b[idx];
    }
}

// CUDA kernel for element-wise multiplication
__global__ void mul_kernel(const float* a, const float* b, float* out, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] * b[idx];
    }
}

// CUDA kernel for matrix multiplication
__global__ void matmul_kernel(const float* a, const float* b, float* out,
                            int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; i++) {
            sum += a[row * k + i] * b[i * n + col];
        }
        out[row * n + col] = sum;
    }
}

// CUDA kernel for ReLU
__global__ void relu_kernel(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] > 0 ? input[idx] : 0;
    }
}

// CUDA kernel for ReLU gradient
__global__ void relu_grad_kernel(const float* input, const float* grad, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] > 0 ? grad[idx] : 0;
    }
}

// CUDA kernel for softmax
__global__ void softmax_kernel(const float* input, float* output, int batch_size, int num_classes) {
    int batch_idx = blockIdx.x;
    int thread_idx = threadIdx.x;
    
    if (batch_idx < batch_size && thread_idx < num_classes) {
        // Find max for numerical stability
        float max_val = input[batch_idx * num_classes];
        for (int i = 1; i < num_classes; i++) {
            max_val = fmaxf(max_val, input[batch_idx * num_classes + i]);
        }
        
        // Compute exp and sum
        float sum = 0.0f;
        for (int i = 0; i < num_classes; i++) {
            float exp_val = expf(input[batch_idx * num_classes + i] - max_val);
            output[batch_idx * num_classes + i] = exp_val;
            sum += exp_val;
        }
        
        // Normalize
        for (int i = 0; i < num_classes; i++) {
            output[batch_idx * num_classes + i] /= sum;
        }
    }
}

// Additional CUDA kernels
__global__ void exp_kernel(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = expf(input[idx]);
    }
}

__global__ void log_kernel(const float* input, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = logf(input[idx]);
    }
}

__global__ void div_kernel(const float* input, float scalar, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] / scalar;
    }
}

__global__ void sub_kernel(const float* a, const float* b, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = a[idx] - b[idx];
    }
}

__global__ void onehot_kernel(const int* indices, size_t num_classes, float* output, size_t batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        int class_idx = indices[idx];
        for (size_t i = 0; i < num_classes; i++) {
            output[idx * num_classes + i] = (i == class_idx) ? 1.0f : 0.0f;
        }
    }
}

__global__ void sum_kernel(const float* input, int axis, bool keepdims, float* output, const size_t* shape, int num_dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < shape[axis]) {
        size_t stride = 1;
        for (int i = axis + 1; i < num_dims; i++) {
            stride *= shape[i];
        }
        
        float sum = 0.0f;
        for (int i = 0; i < shape[axis]; i++) {
            sum += input[i * stride + idx];
        }
        output[idx] = sum;
    }
}

__global__ void max_kernel(const float* input, int axis, bool keepdims, float* output, const size_t* shape, int num_dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < shape[axis]) {
        size_t stride = 1;
        for (int i = axis + 1; i < num_dims; i++) {
            stride *= shape[i];
        }
        
        float max_val = input[idx * stride];
        for (int i = 1; i < shape[axis]; i++) {
            max_val = fmaxf(max_val, input[idx * stride + i]);
        }
        output[idx] = max_val;
    }
}

__global__ void argmax_kernel(const float* input, int axis, int* output, const size_t* shape, int num_dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < shape[axis]) {
        size_t stride = 1;
        for (int i = axis + 1; i < num_dims; i++) {
            stride *= shape[i];
        }
        
        float max_val = input[idx * stride];
        int max_idx = 0;
        for (int i = 1; i < shape[axis]; i++) {
            float val = input[idx * stride + i];
            if (val > max_val) {
                max_val = val;
                max_idx = i;
            }
        }
        output[idx] = max_idx;
    }
}

__global__ void broadcast_kernel(const float* input, float* output, const size_t* input_shape, const size_t* target_shape, int num_dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < target_shape[0]) {
        // Calculate input index based on broadcasting rules
        int input_idx = 0;
        for (int i = 0; i < num_dims; i++) {
            int target_idx = (idx / target_shape[i]) % target_shape[i];
            int input_idx_dim = target_idx % input_shape[i];
            input_idx = input_idx * input_shape[i] + input_idx_dim;
        }
        output[idx] = input[input_idx];
    }
}

__global__ void pow_kernel(const float* input, float power, float* output, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = powf(input[idx], power);
    }
}

// CUDA kernel for transpose
__global__ void transpose_kernel(const float* input, float* output, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m && col < n) {
        output[col * m + row] = input[row * n + col];
    }
}

// C++ wrapper functions for CUDA operations
extern "C" {

void cuda_add(const float* a, const float* b, float* out, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    add_kernel<<<num_blocks, block_size>>>(a, b, out, size);
}

void cuda_mul(const float* a, const float* b, float* out, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    mul_kernel<<<num_blocks, block_size>>>(a, b, out, size);
}

void cuda_matmul(const float* a, const float* b, float* out, int m, int n, int k) {
    dim3 block_dim(16, 16);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, (m + block_dim.y - 1) / block_dim.y);
    matmul_kernel<<<grid_dim, block_dim>>>(a, b, out, m, n, k);
}

void cuda_relu(const float* input, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    relu_kernel<<<num_blocks, block_size>>>(input, output, size);
}

void cuda_relu_grad(const float* input, const float* grad, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    relu_grad_kernel<<<num_blocks, block_size>>>(input, grad, output, size);
}

void cuda_softmax(const float* input, float* output, int batch_size, int num_classes) {
    dim3 block_dim(num_classes);
    dim3 grid_dim(batch_size);
    softmax_kernel<<<grid_dim, block_dim>>>(input, output, batch_size, num_classes);
}

void cuda_exp(const float* input, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    exp_kernel<<<num_blocks, block_size>>>(input, output, size);
}

void cuda_log(const float* input, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    log_kernel<<<num_blocks, block_size>>>(input, output, size);
}

void cuda_div(const float* input, float scalar, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    div_kernel<<<num_blocks, block_size>>>(input, scalar, output, size);
}

void cuda_sub(const float* a, const float* b, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    sub_kernel<<<num_blocks, block_size>>>(a, b, output, size);
}

void cuda_onehot(const int* indices, size_t num_classes, float* output, size_t batch_size) {
    int block_size = 256;
    int num_blocks = (batch_size + block_size - 1) / block_size;
    onehot_kernel<<<num_blocks, block_size>>>(indices, num_classes, output, batch_size);
}

void cuda_sum(const float* input, int axis, bool keepdims, float* output, const size_t* shape, int num_dims) {
    int block_size = 256;
    int num_blocks = (shape[axis] + block_size - 1) / block_size;
    sum_kernel<<<num_blocks, block_size>>>(input, axis, keepdims, output, shape, num_dims);
}

void cuda_max(const float* input, int axis, bool keepdims, float* output, const size_t* shape, int num_dims) {
    int block_size = 256;
    int num_blocks = (shape[axis] + block_size - 1) / block_size;
    max_kernel<<<num_blocks, block_size>>>(input, axis, keepdims, output, shape, num_dims);
}

void cuda_argmax(const float* input, int axis, int* output, const size_t* shape, int num_dims) {
    int block_size = 256;
    int num_blocks = (shape[axis] + block_size - 1) / block_size;
    argmax_kernel<<<num_blocks, block_size>>>(input, axis, output, shape, num_dims);
}

void cuda_broadcast(const float* input, const size_t* input_shape, float* output, const size_t* target_shape, int num_dims) {
    size_t output_size = 1;
    for (int i = 0; i < num_dims; i++) {
        output_size *= target_shape[i];
    }
    int block_size = 256;
    int num_blocks = (output_size + block_size - 1) / block_size;
    broadcast_kernel<<<num_blocks, block_size>>>(input, output, input_shape, target_shape, num_dims);
}

void cuda_pow(const float* input, float power, float* output, size_t size) {
    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    pow_kernel<<<num_blocks, block_size>>>(input, power, output, size);
}

void cuda_transpose(const float* input, float* output, int m, int n) {
    dim3 block_dim(16, 16);
    dim3 grid_dim((n + block_dim.x - 1) / block_dim.x, (m + block_dim.y - 1) / block_dim.y);
    transpose_kernel<<<grid_dim, block_dim>>>(input, output, m, n);
}

} // extern "C" 